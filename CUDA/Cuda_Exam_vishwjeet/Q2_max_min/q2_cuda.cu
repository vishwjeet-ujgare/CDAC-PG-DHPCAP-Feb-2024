#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// #define SIZE 10000000
#define SIZE 1000

#define BLOCK_SIZE 1024

__global__ void findMaxMinKernel(int *arr, int *max, int *min) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= SIZE) return;

    __shared__ int tempMax, tempMin;

    tempMax = arr[idx];
    tempMin = arr[idx];

    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2) {
        if (threadIdx.x + i < blockDim.x) {
            if (arr[idx + i] > tempMax) {
                tempMax = arr[idx + i];
            }
            if (arr[idx + i] < tempMin) {
                tempMin = arr[idx + i];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicMax(max, tempMax);
        atomicMin(min, tempMin);
    }
}

int main() {
    int *arr, *d_arr, *d_max, *d_min;
    int h_max, h_min;

    // Allocate memory on host and device
    arr = (int *)malloc(SIZE * sizeof(int));
    hipMalloc((void **)&d_arr, SIZE * sizeof(int));
    hipMalloc((void **)&d_max, sizeof(int));
    hipMalloc((void **)&d_min, sizeof(int));

    // Initialize array with random values
    for (int i = 0; i < SIZE; i++) {
        arr[i] = rand() % 100;
    }

    // Copy array to device
    hipMemcpy(d_arr, arr, SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Launch kernel
    int numBlocks = (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    findMaxMinKernel<<<numBlocks, BLOCK_SIZE>>>(d_arr, d_max, d_min);

    // Synchronize threads
    hipDeviceSynchronize();

    // Measure execution time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    // Copy result back to host
    hipMemcpy(&h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_min, d_min, sizeof(int), hipMemcpyDeviceToHost);

    printf("Maximum element: %d\n", h_max);
    printf("Minimum element: %d\n", h_min);
    printf("Execution time: %f milliseconds\n", elapsed);

    // Free memory
    free(arr);
    hipFree(d_arr);
    hipFree(d_max);
    hipFree(d_min);

    return 0;
}











