#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N)
        C[tid] = A[tid] + B[tid];
}



void initialData(float *giveArray, int size) {

// //randome float values
//     for (int i = 0; i < size; i++) {
//         giveArray[i] = (float)(rand() & 0xFF) / 10.0f;
//     }

//Or with the values of the irerator as per questions
for (int i = 0; i < size; i++) {
    giveArray[i] = i * 1.0; 
    // printf("%f \n", giveArray[i]);
    }

}

//Sequential additoin of two arrays
void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

int main(int argc, char ** argv) {
    printf("%s Starting...\n", argv[0]);

    // seting up data size of array elements 
    int N = 1000000;

    printf("Array size %d\n", N);

    // allocating host  memory with size_t for N for float
    //to avoid repeatation of counting array size
    size_t nBytes = N * sizeof(float);

    // initializing  pointers or a references for strong gpu address
    float *h_A, *h_B, *hostRef, *gpuRef;

    //assigning memory on host with malloc
    h_A = (float *) malloc(nBytes);
    h_B = (float *) malloc(nBytes);
    hostRef = (float *) malloc(nBytes);
    gpuRef = (float *) malloc(nBytes);

    //Initilize array A and B with randome float numbers or integer as mentioned in the questions
    initialData(h_A, N);
    initialData(h_B, N);

   // sets a block of memory to a specified value. 

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // Performing addition on Host and storing that in hostRef
    auto start_time = std::chrono::high_resolution_clock::now();
    sumArraysOnHost(h_A, h_B, hostRef, N);
    auto end_time = std::chrono::high_resolution_clock::now();


    //measurint time for cpu
    auto duration_ns = std::chrono::duration_cast < std::chrono::nanoseconds > (end_time - start_time).count();
    double seconds_cpu = duration_ns / 1000000000.0;
    std::cout << "Time taken by CPU : " << seconds_cpu << " seconds" << std::endl;


    //code for GPU

    // cudamalloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float **) &d_A, nBytes);
    hipMalloc((float **) &d_B, nBytes);
    hipMalloc((float **) &d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // calling kernel funtion
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sumArraysOnGPU<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds_device=0 ;
    hipEventElapsedTime(&milliseconds_device, start, stop);
    printf("Time taken by device(GPU): %f seconds\n", milliseconds_device/1000);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}
