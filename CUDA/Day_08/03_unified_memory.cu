#include<hip/hip_runtime.h>

#include<stab.h>
#include <stdio.h>
#include <iostream>

__global__ void printValue(int* data){
    int tid=threadIdx.x+blockIdx.x * blockDim.x;

    //Access managed memory directly from the GPU
    printf("GPU THread %d : value = %d \n",tid,data[tid]);
}

int main(){

    const int N =10;

    //Allocate managed memory
    int* data;
    hipMallocManaged(&data,N*sizeof(int));

    //Initialize data on the CPU
    for (int i =0;i<N;++i){
        data[i]=i*2;
    }

    //Launch GPU kernel to print values
    printValue<<<1,N>>>(data);
    hipDeviceSynchronize();//Ensure GPU kernel completes

    //Access managed directly from the CPU
    for(int i=0;i<N;++i){
        printf("CPU : value = %d \n",data[i]);
    }

    //Free managed memory 
    hipFree(data);

    return 0;
}