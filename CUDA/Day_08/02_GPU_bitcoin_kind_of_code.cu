// Dummy "Bitcoin mining" - GPU code
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>

#define TARGET_DIFFICULTY 1000000000

__device__ uint32_t calculateHash(uint32_t nonce) {
    return nonce; // Simplified hash calculation

}

__global__ void mineBitcoin(uint32_t* nonce) {
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t hash;
    do {
        hash = calculateHash(*nonce);
        (*nonce)++;
    }while (hash < TARGET_DIFFICULTY);
}

int main() {
    uint32_t* d_nonce;
    hipMalloc(&d_nonce, sizeof(uint32_t));

    uint32_t initialNonce = 0;
    hipMemcpy(d_nonce, &initialNonce, sizeof(uint32_t), hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int numBlocks = (TARGET_DIFFICULTY + threadsPerBlock - 1) / threadsPerBlock;

    auto start_time = std::chrono::high_resolution_clock::now();
    mineBitcoin <<<numBlocks, threadsPerBlock>>> (d_nonce);
    auto end_time = std::chrono::high_resolution_clock::now();

    auto duration_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count();
    hipDeviceSynchronize();
    std::cout<<"Time taken by GPU   : " << duration_ns << " nanoseconds"<< std::endl;

    hipFree(d_nonce);
    return 0;
}