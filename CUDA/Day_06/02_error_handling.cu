#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA kernel to add two arrays element-wise with more thread details

__global__ void addArrays(const int* a, const int* b, int* result, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    //thread details
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;

    //int threadsPerBlock = blockDim.x;
    int totalThreads = blockDim.x * gridDim.x;

    //check if the thread index is withim the valid range
    if(idx < size){
        printf("Thread %d (Block %d, Thread in Block %d, Total Thread %d): Adding %d + %d = %d\n", idx, blockId, threadId, totalThreads, a[idx], b[idx], a[idx] + b[idx]);
        result[idx] = a[idx] + b[idx];

    
    }
}
int main(){
    const int arraySize = 100;

    //Host (CPU) data
    int hostArray1[arraySize];
    int hostArray2[arraySize];
    int hostResultArray[arraySize];

    //Generate random integer numbers for the host arrays
    srand((unsigned)time(NULL));
    for(int i = 0; i < arraySize; i++){
        hostArray1[i]= rand() %100; //Random number between 0 to 99
        hostArray2[i]= rand() %100; //Random number between 0 to 99
    }

    //Device (GPU) data
    int* deviceArray1;
    int* deviceArray2;
    int* deviceResultArray;

    //Allocate device memory
    hipError_t cudaStatus;

    // cudaStatus = cudaMalloc((void**)&deviceArray1, arraySize * sizeof(int));
    cudaStatus = hipMalloc((void**)&deviceArray1, 0);//error will occure
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "hipMalloc for deviceArray1 failed : %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);

    }
    cudaStatus = hipMalloc((void**)&deviceArray2, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "hipMalloc for deviceArray2 failed : %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    cudaStatus = hipMalloc((void**)&deviceResultArray, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "hipMalloc for deviceResultArray failed : %s\n", hipGetErrorString(cudaStatus));
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        exit(EXIT_FAILURE);
    }

    //copy data from cpu to gpu
    // on purpose
    /*
    cudaStatus = cudaMemcpy(deviceArray1, hostArray1, arraySize * sizeof(int),cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess){
        fprintf(stderr, "cudaMemcpy (host to device) for deviceArray1 failed : %s\n", cudaGetErrorString(cudaStatus));
        cudaFree(deviceArray1);
        cudaFree(deviceArray2);
        cudaFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }

    */
   //checking for error
        cudaStatus = hipMemcpy(deviceArray1, hostArray1, arraySize * sizeof(int),hipMemcpyHostToDevice);

     cudaStatus = hipMemcpy(deviceArray2, hostArray2,  arraySize *  sizeof(int),hipMemcpyHostToDevice);
     if (cudaStatus != hipSuccess){
        fprintf(stderr, "hipMemcpy (host to device) for deviceArray2 failed : %s\n", hipGetErrorString(cudaStatus));
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }

    //Launce the kernel to add arrays on the device
    addArrays <<<1, arraySize >>>(deviceArray1, deviceArray2, deviceResultArray, arraySize);

    //Synchronize to ensure kernel execution is completed before proceeding
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "hipDeviceSynchronize failed : %s\n", hipGetErrorString(cudaStatus));
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }

    //copy the result data from GPU to CPU
    cudaStatus = hipMemcpy(hostResultArray, deviceResultArray, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "hipMemcpy (device to host) for hostResultArray failed : %s\n", hipGetErrorString(cudaStatus));
        hipFree(deviceArray1);
        hipFree(deviceArray2);
        hipFree(deviceResultArray);
        exit(EXIT_FAILURE);
    }

    //Display results
    printf("Array 1 : ");
    for(int i = 0; i < arraySize; i++){
        printf("%d ", hostArray1[i]);

    }
    printf("\n");
    printf("Array 2 : ");
    for(int i = 0; i < arraySize; i++){
        printf("%d ", hostArray2[i]);
    }
    printf("\n");

    printf("Result Array : ");
    for(int i = 0; i < arraySize; i++){
        printf("%d ", hostResultArray[i]);
    }
    printf("\n");

    //Free allocateed memory on GPU
    hipFree(deviceArray1);
    hipFree(deviceArray2);
    hipFree(deviceResultArray);

    return 0;
}