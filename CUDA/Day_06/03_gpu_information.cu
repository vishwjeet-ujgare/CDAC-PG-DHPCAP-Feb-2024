#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>

int main(){
    int deviceCount;

    hipGetDeviceCount(&deviceCount);

    if(deviceCount==0){
        printf("No cuda device found.\n");
        return 1;
    }


    for (int device=0;device<deviceCount;++device){
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp , device);


        printf("              Device %d : %s\n",device,deviceProp.name);
        printf("     Coputer Capability : %d.%d\n",deviceProp.major,deviceProp.minor);
        printf("    Total Global Memory : %lu\n",(unsigned long)deviceProp.totalGlobalMem);
        printf("Shared memory per BLock : %lu bytes\n",(unsigned long) deviceProp.sharedMemPerBlock);
        
        printf("              wrap Size : %d\n",deviceProp.warpSize);
        printf("  Max Threads Per Block : %d\n",deviceProp.maxThreadsPerBlock);
        printf(" Max Threads Diamension : (%d, %d,%d)\n",deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2] );
        printf("          Max Grid Size : (%d, %d,%d)\n",deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxGridSize[2]);

        printf("             Clock Rate : %d kHz\n",deviceProp.clockRate);
        printf("      Memory Clock Rate : %d kHz\n",deviceProp.memoryClockRate);
        printf("       Memory Bus width : %d bits\n",deviceProp.memoryBusWidth);
        printf("          L2 Cache size : %d bytes \n",deviceProp.l2CacheSize);
        printf("   Constant memory size : %lu bytyes \n",(unsigned long) deviceProp.totalConstMem);

        printf("      Texture Alignment : %lu bytyes \n",(unsigned long) deviceProp.textureAlignment);
        printf("\n");
       

    }
     return 0;
}