
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <iostream>

#define NUM_CLASSES 3
#define NUM_HOUSE_TYPES 5 // assume 5 house types

__global__ void countHouseTypes(int *houseTypeArray, int *houseTypeCounts, int totalRecords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalRecords) {
        atomicAdd(&houseTypeCounts[houseTypeArray[idx]], 1);
    }
}

__global__ void countGenders(int *genderArray, int *maleCount, int *femaleCount, int totalRecords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalRecords) {
        if (genderArray[idx] == 0) {
            atomicAdd(maleCount, 1);
        } else {
            atomicAdd(femaleCount, 1);
        }
    }
}

int main() {
    // Read the CSV files
    FILE *file1 = fopen("/home/user3/CUDA/ASSIGNMENT/pune.csv", "r");
    FILE *file2 = fopen("/home/user3/CUDA/ASSIGNMENT/mumbai.csv", "r");
    FILE *file3 = fopen("/home/user3/CUDA/ASSIGNMENT/mumbai-suburuban.csv", "r");

    if (file1 == NULL || file2 == NULL || file3 == NULL) {
        fprintf(stderr, "Error opening file.\n");
        return 1;
    }

    printf("Files opened successfully.\n");

    int numRecords1 = 0, numRecords2 = 0, numRecords3 = 0;

    // Count the number of records in each file
    while (!feof(file1)) {
        char buffer[265];
        if (fgets(buffer, sizeof(buffer), file1) == NULL) break;
        numRecords1++;
    }

    while (!feof(file2)) {
        char buffer[265];
        if (fgets(buffer, sizeof(buffer), file2) == NULL) break;
        numRecords2++;
    }

    while (!feof(file3)) {
        char buffer[265];
        if (fgets(buffer, sizeof(buffer), file3) == NULL) break;
        numRecords3++;
    }

    int totalRecords = numRecords1 + numRecords2 + numRecords3;

    // Allocate memory on CPU
    int *houseTypeArray = (int *) malloc(totalRecords * sizeof(int));
    int *genderArray = (int *) malloc(totalRecords * sizeof(int));

    // Reset the file pointers to the beginning
    rewind(file1);
    rewind(file2);
    rewind(file3);

    int i = 0;
    char line[256];
    while (fgets(line, sizeof(line), file1)!= NULL) {
        char *token = strtok(line, ",");
        // assume house type is the second column
        if (strcmp(token, "Apartment") == 0) {
            houseTypeArray[i] = 0; // apartment
        } else if (strcmp(token, "House") == 0) {
            houseTypeArray[i] = 1; // house
        } else if (strcmp(token, "Flat") == 0) {
            houseTypeArray[i] = 2; // flat
        } else if (strcmp(token, "Bungalow") == 0) {
            houseTypeArray[i] = 3; // bungalow
        } else if (strcmp(token, "Other") == 0) {
            houseTypeArray[i] = 4; // other
        }

        token = strtok(NULL, ",");
        // assume gender is the third column
        if (strcmp(token, "Male") == 0) {
            genderArray[i] = 0; // male
        } else if (strcmp(token, "Female") == 0) {
            genderArray[i] = 1; // female
        }
        i++;
    }

    i = numRecords1;
    while (fgets(line, sizeof(line), file2)!= NULL) {
        char *token = strtok(line, ",");
        // assume house type is the second column
        if (strcmp(token, "Apartment") == 0) {
            houseTypeArray[i] = 0; // apartment
        } else if (strcmp(token, "House") == 0) {
            houseTypeArray[i] = 1; // house
        } else if (strcmp(token, "Flat") == 0) {
            houseTypeArray[i] = 2; // flat
        } else if (strcmp(token, "Bungalow") == 0) {
            houseTypeArray[i] = 3; // bungalow
        } else if (strcmp(token, "Other") == 0) {
           houseTypeArray[i] = 4; // other
        }

        token = strtok(NULL, ",");
        // assume gender is the third column
        if (strcmp(token, "Male") == 0) {
            genderArray[i] = 0; // male
        } else if (strcmp(token, "Female") == 0) {
            genderArray[i] = 1; // female
        }
        i++;
    }

    i = numRecords1 + numRecords2;
    while (fgets(line, sizeof(line), file3)!= NULL) {
        char *token = strtok(line, ",");
        // assume house type is the second column
        if (strcmp(token, "Apartment") == 0) {
            houseTypeArray[i] = 0; // apartment
        } else if (strcmp(token, "House") == 0) {
            houseTypeArray[i] = 1; // house
        } else if (strcmp(token, "Flat") == 0) {
            houseTypeArray[i] = 2; // flat
        } else if (strcmp(token, "Bungalow") == 0) {
            houseTypeArray[i] = 3; // bungalow
        } else if (strcmp(token, "Other") == 0) {
            houseTypeArray[i] = 4; // other
        }

        token = strtok(NULL, ",");
        // assume gender is the third column
        if (strcmp(token, "Male") == 0) {
            genderArray[i] = 0; // male
        } else if (strcmp(token, "Female") == 0) {
            genderArray[i] = 1; // female
        }
        i++;
    }

    fclose(file1);
    fclose(file2);
    fclose(file3);

    // Allocate memory on GPU
    int *d_houseTypeArray;
    hipMalloc((void **)&d_houseTypeArray, totalRecords * sizeof(int));
    hipMemcpy(d_houseTypeArray, houseTypeArray, totalRecords * sizeof(int), hipMemcpyHostToDevice);

    int *d_houseTypeCounts;
    hipMalloc((void **)&d_houseTypeCounts, NUM_HOUSE_TYPES * sizeof(int));
    hipMemset(d_houseTypeCounts, 0, NUM_HOUSE_TYPES * sizeof(int));

    int *d_genderArray;
    hipMalloc((void **)&d_genderArray, totalRecords * sizeof(int));
    hipMemcpy(d_genderArray, genderArray, totalRecords * sizeof(int), hipMemcpyHostToDevice);

    int *d_maleCount;
    hipMalloc((void **)&d_maleCount, sizeof(int));
    hipMemset(d_maleCount, 0, sizeof(int));

    int *d_femaleCount;
    hipMalloc((void **)&d_femaleCount, sizeof(int));
    hipMemset(d_femaleCount, 0, sizeof(int));

    // auto cpuStart = std::chrono::high_resolution_clock::now();

    // Calculate the percentage of various house types
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalRecords + threadsPerBlock - 1) / threadsPerBlock;
    countHouseTypes<<<blocksPerGrid, threadsPerBlock>>>(d_houseTypeArray, d_houseTypeCounts, totalRecords);

    // Calculate the ratio of males:females
    countGenders<<<blocksPerGrid, threadsPerBlock>>>(d_genderArray, d_maleCount, d_femaleCount, totalRecords);

    hipDeviceSynchronize();

    // auto cpuEnd = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> cpuElapsed = cpuEnd - cpuStart;
   

    auto gpuStart = std::chrono::high_resolution_clock::now();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    countHouseTypes<<<blocksPerGrid, threadsPerBlock>>>(d_houseTypeArray, d_houseTypeCounts, totalRecords);
    countGenders<<<blocksPerGrid, threadsPerBlock>>>(d_genderArray, d_maleCount, d_femaleCount, totalRecords);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gpuElapsedTime;
    hipEventElapsedTime(&gpuElapsedTime, start, stop);
   

    int houseTypeCounts[NUM_HOUSE_TYPES] = {0, 0, 0, 0, 0};
    hipMemcpy(houseTypeCounts, d_houseTypeCounts, NUM_HOUSE_TYPES * sizeof(int), hipMemcpyDeviceToHost);

    int maleCount, femaleCount;
    hipMemcpy(&maleCount, d_maleCount, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&femaleCount, d_femaleCount, sizeof(int), hipMemcpyDeviceToHost);

    // Free the allocated memory
    free(houseTypeArray);
    free(genderArray);
   hipFree(d_houseTypeArray);
    hipFree(d_houseTypeCounts);
    hipFree(d_genderArray);
    hipFree(d_maleCount);
    hipFree(d_femaleCount);

    // Print the results
    printf("House type percentages:\n");
    for (int i = 0; i < NUM_HOUSE_TYPES; i++) {
        printf("%s: %.2f%%\n", i == 0? "Apartment" :i == 1? "House" : i == 2? "Flat" : i == 3? "Bungalow" : "Other",
               (float) houseTypeCounts[i] / totalRecords * 100);
    }

    printf("Ratio of males:females: %.2f:%.2f\n", (float) maleCount / totalRecords * 100,
           (float) femaleCount / totalRecords * 100);

        printf("GPU time: %.2f seconds\n", gpuElapsedTime);

    return 0;
}