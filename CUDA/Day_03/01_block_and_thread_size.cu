#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add_arrays(int *c , const int  *a, const int *b , int size){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
   
    if(i<size){
        c[i]=a[i]+b[i];
    }
}


int main(){
    const int size=90000;

    int a[size],b[size];

    for (int i =0;i<size;i++)
    {
         a[i]=i;
         b[i]=i;
    }

    int *d_c;

    //Allocate memory on the device for array c
    hipMalloc((void**)&d_c,size * sizeof(int));

    //Copy arrays a and b to the device
    int *d_a,*d_b;

    hipMalloc((void**)&d_a , size*sizeof(int));
    hipMalloc((void**)&d_b , size*sizeof(int));
    
    hipMemcpy(d_a,a,size * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size * sizeof(int),hipMemcpyHostToDevice);

    int threadPerBlock=512;
    int blockPerGrid=(size+threadPerBlock-1)/threadPerBlock;
    // printf("Block per grid : %d \n",blockPerGrid);
   

    // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    add_arrays<<<blockPerGrid,threadPerBlock>>>(d_c,d_a,d_b,size);
    hipDeviceSynchronize();


// Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
   
    //copy thre result back from the device
    int *c =(int*)malloc(size * sizeof(int));

    hipMemcpy(c,d_c,size*sizeof(int),hipMemcpyDeviceToHost);

    //print the reslult
    for (int i=0;i<size;i++){
        printf("%d).%d \n",i,c[i]);
    }

    printf("\n");

  
    // Print time taken by GPU
    printf("Time taken by GPU : %f milliseconds\n", milliseconds);
    //free me
    hipFree(c);
    hipFree(d_a);
    hipFree(d_c);
    hipFree(d_b);


}