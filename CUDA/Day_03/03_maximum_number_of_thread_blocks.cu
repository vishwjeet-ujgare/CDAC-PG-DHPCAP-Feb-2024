//finding maximum number of threads ans blocks on out GPU ...

#include<iostream>
#include<hip/hip_runtime.h>

int main(){
    int maxThreadsBlock,maxBlocks;

    hipDeviceGetAttribute(&maxThreadsBlock,hipDeviceAttributeMaxThreadsPerBlock,0);
    hipDeviceGetAttribute(&maxBlocks,hipDeviceAttributeMaxGridDimX,0);

    std::cout<<"Maximum Threads per Block: "<<maxThreadsBlock<<std::endl;
    std::cout<<"Maximum  Block: "<<maxBlocks<<std::endl;
    return 0 ;
    
}