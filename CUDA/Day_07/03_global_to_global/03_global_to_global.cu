#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void childKernel(){
    printf("Hello from child kernel function\n");
}


__global__ void parentKernel(){
    childKernel<<<1,1>>>();
    hipDeviceSynchronize();//wair for the child to comple
    printf("Hello from parent kernel function\n");
}

int main(){
    parentKernel<<<1,1>>>();
    hipDeviceSynchronize();//wait for the parent to complete

    return 0;
}

// nvcc 03_global_to_global.cu -rdc=true-