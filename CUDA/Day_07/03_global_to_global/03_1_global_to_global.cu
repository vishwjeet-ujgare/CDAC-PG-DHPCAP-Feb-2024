#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void grandChildKernel(){
    printf("Hello from grand child kernel function\n");
}


__global__ void childKernel(){
    grandChildKernel<<<1,1>>>();
    hipDeviceSynchronize();//wair for the child to comple
    printf("Hello from child kernel function\n");
}


__global__ void parentKernel(){
    childKernel<<<1,1>>>();
    // hipDeviceSynchronize();//wair for the child to comple
    printf("Hello from parent kernel function\n");
}

int main(){
    parentKernel<<<1,1>>>();
    hipDeviceSynchronize();//wait for the parent to complete

    return 0;
}

// nvcc 03_global_to_global.cu -rdc=true-