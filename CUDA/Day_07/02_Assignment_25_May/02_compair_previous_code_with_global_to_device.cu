// Assignment (25 May):Dyanamic Parallelism - 
    // Maintain a copy of the original code. Modify the code to have the square functinality inside the doubleValue function itself . Have 10000000 elements , instead of 10 in both versions. Compare the performance of the two versions by adding timing code.

#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void doubleValues(int* data, int size){
    int i= blockIdx.x * blockDim.x + threadIdx.x;
    if(i<size){
        int value =data[i];
        data [i]=(value * value) * 2;
    }

}

int main(){
    //Allocate memory on host and device
    int size=10000000;

    int* data_host=new int[size];
    int* data_device;
    hipMalloc(&data_device, size * sizeof(int));

    //Initialize data on host
    for (int i=0;i<size;++i){
        data_host[i]=i;
    }

    //Copy data to device
    hipMemcpy(data_device,data_host,size * sizeof(int),hipMemcpyHostToDevice);


  //Timing code
    float gpuElapsed=0.0;
    hipEvent_t start,stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    //launch kernel
    int threadsPerBlock=256;

    //Record start time
    hipEventRecord(start);
    doubleValues<<<(size + threadsPerBlock -1)/threadsPerBlock , threadsPerBlock>>>(data_device, size);

    //wait for kernel to finisj 
    hipDeviceSynchronize();

    //record end time
    hipEventRecord(stop);
        hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuElapsed,start,stop);


    //copy data back from device
    hipMemcpy(data_host,data_device,size * sizeof(int),hipMemcpyDeviceToHost);

    //print results
    for (int i =0;i<size;++i){
        // printf("data[%d]=%d\n", i , data_host[i]);
    }

    //printing time 
    printf("GPU Processing time : %0.10f ms\n",gpuElapsed);

    //Free memory
    hipFree(data_device);
    delete[] data_host;

    return 0;

}