#include<hip/hip_runtime.h>
#include<stdio.h>
// #include<cuda_runtime.h>

__device__ int square(int a){
    // printf("Thread(%d,%d) -  squaring values \n",blockIdx.x,threadIdx.x);

    return a*a;
}


__global__ void doubleValues(int* data, int size){
    int i= blockIdx.x * blockDim.x + threadIdx.x;
    if(i<size){
        int value =data[i];
        //call square function and print thread ID
        int squared_value=square(value);
        // printf("Thread (%d , %d )-doubling squared value \n",blockIdx.x,threadIdx.x);
        data [i]=squared_value * 2;
    }

}

int main(){
    //Allocate memory on host and device
    int size=100000000;

    int* data_host=new int[size];
    int* data_device;

    hipMalloc(&data_device, size * sizeof(int));

    //Initialize data on host
    for (int i=0;i<size;++i){
        data_host[i]=i;
    }

    //Copy data to device
    hipMemcpy(data_device,data_host,size * sizeof(int),hipMemcpyHostToDevice);

    //Time code
    float gpuElapsed=0.0;
    hipEvent_t start,stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    //launch kernel
    int threadsPerBlock=256;

    hipEventRecord(start);
    doubleValues<<<(size + threadsPerBlock -1)/threadsPerBlock , threadsPerBlock>>>(data_device, size);

    //wait for kernel to finisj 
    hipDeviceSynchronize();


    //Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuElapsed,start,stop);


    //copy data back from device
    hipMemcpy(data_host,data_device,size * sizeof(int),hipMemcpyDeviceToHost);

    //print results
    for (int i =0;i<size;++i){
        // printf("data[%d]=%d\n", i , data_host[i]);
    }

    //printing time 
    printf("GPU Processing time : %0.10f ms\n",gpuElapsed);

    //Free memory
    hipFree(data_device);
    delete[] data_host;

    return 0;

}