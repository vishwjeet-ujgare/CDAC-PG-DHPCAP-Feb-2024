#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void kernel(void){
    printf("Hello from GPU\n");
}

void cpu_print(void)
{
    printf("Hello form CPU\n");
}

int main()
{
    // kernel <<<1,1>>>();
    // kernel <<<1,10>>>();
    kernel <<<2,2>>>();
    
    hipDeviceSynchronize();

    printf("----------------\n");
    cpu_print();
    return 0;
}