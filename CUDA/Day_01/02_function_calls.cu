#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernal(void){

    printf("Hello from GPU\n");

}


void cpu_print(void){
    printf("Hello from CPU\n");
}


int main(){

    kernal <<<1,1>>>();
    kernal <<<1,1>>>();
    kernal <<<1,1>>>();
//ones cpu done there work it terminated and does not wait for GPU work do be completed

    cpu_print();
    cpu_print();
    cpu_print();

    return 0;
}