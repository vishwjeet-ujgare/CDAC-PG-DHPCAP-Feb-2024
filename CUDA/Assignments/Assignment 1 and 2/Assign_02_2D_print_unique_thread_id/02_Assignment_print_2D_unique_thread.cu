#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 2
int rows = 2;
int cols = 2;
dim3 dimBlock(N,N);
dim3 dimGrid(rows + N - 1 / N, cols + N - 1 / N);

__global__ void displayGlobalId()
{
    int tidX = threadIdx.x + blockIdx.x * blockDim.x;
    int tidY = threadIdx.y + blockIdx.y * blockDim.y;
    int tBlocks = N*N;
    int Gid = tidX * tBlocks + tidY;
    printf("Global TID: %d\n", Gid);
}

int main()
{
    displayGlobalId<<<dimGrid,dimBlock>>>();
    hipDeviceSynchronize();
    return 0;
}