//GPU

#include<stdio.h>
#include<hip/hip_runtime.h>
#define NUM 500000

//CUDA kernel function for printing prime number
__global__ void printPrimeNumbers(){
    int threadID=blockIdx.x * blockDim.x + threadIdx.x;

if(threadID>2 && threadID<=NUM)
{ 
  int flag=1;//it is prime number
    
    for (int i = 2; i < threadID; i++)
    {
        if(threadID%i==0){
            flag=0;//it it not a prime number
            break;
        }
    }

    if(flag)
    {
        // printf("%d ",threadID);
    }           
}

}



int main()
{
    // create host var
    int host_fromNum = 2;
    int host_toNum =500000;


    //Declare for Device variables/pointers
    int *device_fromNum;
    int *device_toNum;

    //Allocate memory on the device/host
    hipMalloc((void**)&device_fromNum, sizeof(int));
    hipMalloc((void**)&device_toNum, sizeof(int));

    //copy data from host to device
    hipMemcpy(device_fromNum,&host_fromNum,sizeof(int),hipMemcpyHostToDevice);//cudaMemcpyHostToDevice
    hipMemcpy(device_toNum,&host_toNum,sizeof(int),hipMemcpyHostToDevice);//

    int threadPerBlock=512;
    int blockPerGrid=(host_toNum+threadPerBlock-1)/threadPerBlock;

   // Start timing GPU execution
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    printPrimeNumbers <<<blockPerGrid,threadPerBlock>>>();
    hipDeviceSynchronize();

// Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print time taken by GPU
    printf("\nTime taken by GPU : %f milliseconds\n", milliseconds);

    //free alloacted memory
    hipFree(device_fromNum);
    hipFree(device_toNum);


}    