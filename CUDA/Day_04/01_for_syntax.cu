#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vectorAddition_kernel(int* d_a,int* d_b,int* d_c,int N){
    int tid=threadIdx.x;
    
    if(tid<N){
        d_c[tid]=d_a[tid]+d_b[tid];
    }
}


int main()
{

    int N=8;
    int* h_a,*h_b,*h_c;//host variables
    int* d_a,*d_b,*d_c;//Device variables

    //Allocate memory for host variables
    h_a=(int*)malloc(N * sizeof(int));
    h_b=(int*)malloc(N * sizeof(int));
    h_c=(int*)malloc(N * sizeof(int));


    //Allocate memory for device variables
    hipMalloc((void**)&d_a,N*sizeof(int));
    hipMalloc((void**)&d_b,N*sizeof(int));
    hipMalloc((void**)&d_c,N*sizeof(int));

    //Initialize host variable
    for(int i=0;i<N;i++){
        h_a[i]=2;
        h_b[i]=2;
        h_c[i]=0;
    }

    //copy host variables to device
    hipMemcpy(d_a,h_a,N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,N * sizeof(int),hipMemcpyHostToDevice);

    //Launch kernel function
    int blockSize=1;
    // int numThreads=N;
    int numThreads=4;//to see what

    vectorAddition_kernel<<<blockSize,numThreads>>>(d_a,d_b,d_c,N);

    //copy result back to host
    hipMemcpy(h_c,d_c,N * sizeof(int),hipMemcpyDeviceToHost);

    //Display results
    printf("Result : ");

    for(int i=0;i<N;i++){
        printf("%d ",h_c[i]);
    }

    printf("\n");

     //free  device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    
}