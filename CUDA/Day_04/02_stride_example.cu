#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vectorAddition_kernel(int* d_a,int* d_b,int* d_c,int N){
    int tid=threadIdx.x;
    int i;
    for(i = tid; i < N; i += blockDim.x){
        d_c[i]=d_a[i]+d_b[i];
        printf("TID :%d = %d \n",tid,d_c[i]);
    }
}


int main()
{

    // int N=8;
    int N=12;
    int* h_a,*h_b,*h_c;//host variables
    int* d_a,*d_b,*d_c;//Device variables

    //Allocate memory for host variables
    h_a=(int*)malloc(N * sizeof(int));
    h_b=(int*)malloc(N * sizeof(int));
    h_c=(int*)malloc(N * sizeof(int));


    //Allocate memory for device variables
    hipMalloc((void**)&d_a,N*sizeof(int));
    hipMalloc((void**)&d_b,N*sizeof(int));
    hipMalloc((void**)&d_c,N*sizeof(int));

    //Initialize host variable
    for(int i=0;i<N;i++){
        h_a[i]=2;
        h_b[i]=2;
        h_c[i]=0;
    }

    //copy host variables to device
    hipMemcpy(d_a,h_a,N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,N * sizeof(int),hipMemcpyHostToDevice);

    //Launch kernel function
    int blockSize=1;
    // int numThreads=N;
    int numThreads=4;//to see what

    vectorAddition_kernel<<<blockSize,numThreads>>>(d_a,d_b,d_c,N);

    //copy result back to host
    hipMemcpy(h_c,d_c,N * sizeof(int),hipMemcpyDeviceToHost);

    //Display results
    printf("Result : ");

    for(int i=0;i<N;i++){
        printf("%d ",h_c[i]);
    }

    printf("\n");

     //free  device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    
}